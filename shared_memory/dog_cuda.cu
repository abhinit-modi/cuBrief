#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/cudaarithm.hpp>
#include <opencv2/opencv.hpp>
#include <opencv2/features2d.hpp>
#include <iomanip>
#include <vector>
#include <cstdio>
#include <ctime>
#include "opencv2/xfeatures2d.hpp"
#include <opencv2/xfeatures2d/nonfree.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "CycleTimer.h"
#include "filter.h"
#include "filter.cu_incl"

//#define DEBUG

#ifdef DEBUG
/* When debugging is enabled, these form aliases to useful functions */
#define dbg_printf(...) printf(__VA_ARGS__); 
#else
/* When debugging is disnabled, no code gets generated for these */
#define dbg_printf(...)
#endif


using namespace cv;
using namespace cv::cuda;
using namespace std;

#define MAX_IMG_SZ 4096 * 4096 *2

#define MAX_PYRAMIDS 5

#define BSZ 32 

/* Global memory for keypoints */
const int max_kp = 12000;

__device__ int kp = 0;
__device__ int kp_x[max_kp];
__device__ int kp_y[max_kp];

__device__ float d[MAX_PYRAMIDS][MAX_IMG_SZ];
__device__ float s[MAX_PYRAMIDS + 1][MAX_IMG_SZ];

__device__ void img_to_s(float *img, float *s, int w, int h, int x, int y)
{
    for (int yy = -1; yy <= 1; yy++)
        for (int xx = -1; xx <= 1; xx++)
        {
            int x_i = x +xx;
            int y_i = y + yy;

            if (x_i < 0 || y_i < 0 || x_i >= w || y_i >= h)
                s[ (yy + 1) * 3 + (xx +1)] = 0;
            else
                s[ (yy + 1) * 3 + (xx +1)] = img[(y_i * w) + x_i];
        }

}

__global__ void vertical_blur( float* img,
                            int w,
                            int h,
                            filter_bank fb)
{

    /*2D Index of current thread */
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int padding = fb.bank[fb.n - 1].k ;
    const int cache_width = blockDim.x + padding;
    const int cache_height = blockDim.y;

    __shared__ float cache_image[(BSZ+25)*BSZ+25];

    const int center = 3*(1) + 1;
    const int levels = fb.n;
    __shared__ float fy[50];
    __shared__ float fx[50];

    /* Out of bounds pixel */
//    if (x != 0 || y != 0)
    if (x >= w || y >= h)
        return;
      
    /* Collect shared cache_image */
    cache_image[(threadIdx.y * cache_width) + threadIdx.x + padding/2 ] = img[(y*w) + x];

    /* Borders are collected by first threads */
    if (threadIdx.x < padding/2 && (x- padding/2)  >= 0)
        cache_image[(threadIdx.y * cache_width) + threadIdx.x] = img[(y*w) + x - padding/2];

    /* And last threads */
    if (threadIdx.x > cache_width - padding/2 && (x + padding/2) < w )
        cache_image[(threadIdx.y * cache_width) + threadIdx.x] = img[(y*w) + x + padding/2];

    for (int i = 0; i < levels; i ++)
    {
        int k = fb.bank[i].k;

        linear_filter lf;
        lf.hy = fy;
        lf.hx = fx;
        lf.k = fb.bank[i].k;

        int lin_index = threadIdx.y*blockDim.x + threadIdx.x;
        square_to_linear(fb.bank[i], &lf, lin_index);
        __syncthreads();
        float response = s[i][(y*w) + x] = get_filter_response_horizontal(cache_image, cache_width, cache_height, lf, threadIdx.x, threadIdx.y);
//float response = s[i][(y*w) + x] = get_filter_response_horizontal(img, w, h, lf, x, y);
//        if (x == 100 && y == 100)
//            printf("Response 100 100 horizontal is %f \n", response);
//        printf("Testing filter %d by %d \n",k,k);       
//        for (int m = 0;m < k; m++)
//            printf("%f ",lf.hx[m]);

        s[i][(y*w) + x] = response;
    }
    //printf("Test1 \n");
}


__global__ void horizontal_blur( float* img,
                            int w,
                            int h,
                            filter_bank fb)
{

    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    const int center = 3*(1) + 1;
    const int levels = fb.n - 1;
    float fx[50];
    float fy[50];
    float response = 0;

    float sk, sk_1;

    if (x >= w || y >= h)
        return;


    int k = fb.bank[0].k;

    linear_filter lf;
    lf.hx = fx;
    lf.hy = fy;

    lf.k = fb.bank[0].k;
    int lin_index = threadIdx.y*blockDim.x + threadIdx.x;
    square_to_linear(fb.bank[0], &lf, lin_index);
    // square_to_linear(fb.bank[0], &lf);
    __syncthreads();
    sk = get_filter_response_vertical(s[0], w, h, lf, x, y);
    
//    if (x == 100 && y == 100)
//        printf("Response 100 100 horizontal is %f \n", sk);

    for (int i = 0; i < levels; i ++)
    {
        k = fb.bank[i+1].k;

        linear_filter lf;
        lf.hx = fx;
        lf.hy = fy;
    
        lf.k = k;
        
        int lin_index = threadIdx.y*blockDim.x + threadIdx.x;
        square_to_linear(fb.bank[i+1], &lf, lin_index);
        // square_to_linear(fb.bank[i+1], &lf);
        __syncthreads();
        sk_1 = get_filter_response_vertical(s[i+1], w, h, lf, x, y);

        d[i][(y*w) + x] = sk - sk_1;

        sk = sk_1;
 //           if (x == 100 && y == 100)
 //       printf("Response 100 100 horizontal is %f \n", sk);

    }
    //printf("Test 2\n");
}


__global__ void DoG_Pyramid( float* img,
                            int w,
                            int h,
                            filter_bank fb)
{

    /*2D Index of current thread */
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    const int center = 3*(1) + 1;
    const int levels = fb.n - 1;
    float sk, sk_1;

    /* Out of bounds pixel */
    if (x >= w || y >= h)
        return;

    sk = get_filter_response(img, w, h, fb.bank[0], x, y);

    for (int i = 0; i < levels; i ++)
    {
        //sk = get_filter_response(img, w, h, fb.bank[i], x, y);
        sk_1 = get_filter_response(img, w, h, fb.bank[i+1], x, y);        
        d[i][(y*w) + x] = sk - sk_1; 
        sk = sk_1;
    }

}
__global__ void DoG_Kernel( float* img,
                            int w,
                            int h,
                            filter_bank fb)
{
    /*2D Index of current thread */
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    const int center = 3*(1) + 1;
    const int levels = fb.n - 1;
    const float th_c = 7.0;
    const float th_r = 12.0;

    /* Out of bounds pixel */
    if (x >= w || y >= h)
        return;

    /* Pixels in the border */
    if ( (x >= w-4) || (y >= h-4) || x < 4 || y < 4)
        return;
 
    /* DoG first levels */

    /* D(k-1), D(k), D(k+1) */
    float d_1k[9], dk[9], dk_1[9];
  
    /* Regsiters to calculate Hessian of DoG */
    float dh[25], sh_1[25], sh[25];
 
    /* Compute D(k) and D(k+1) for first level */

    for (int i = 0; i < levels; i++)
    {        
        float current = d[i][(y*w) + x];
        bool ismax = true;
        bool ismin = true;

     /* If threshold test fails go to next iteration */
        if (fabs(current) < th_c)
            continue;
 
        img_to_s(d[i], dk, w, h, x, y);

        /* Current layer */
        ismax = ismax && is_max(dk, current);
        ismin = ismin && is_min(dk, current);

        if (!ismax && !ismin)
            continue;

        /* Layer below */
        if (i != levels - 1)
        {
            img_to_s(d[i+1], dk_1, w, h, x, y);

            ismax = ismax && is_max(dk_1, current);
            ismin = ismin && is_min(dk_1, current);

            if (!ismax && !ismin)
                continue;
        }

        /* Layer above */
        if (i != 0)
        {
            img_to_s(d[i-1], d_1k, w, h, x, y);
            ismax = ismax && is_max(d_1k, current);
            ismin = ismin && is_min(d_1k, current);

            if (!ismax && !ismin)
                continue;
        }


        float R = get_curvature(d[i], w, h, x, y);
    
        if (R > th_r)
            break;
        /* Atomically increase the number of keypoints
           and add the new found keypoint 
        */

        int idx = atomicAdd(&kp, 1);

        kp_x[idx] = x;
        kp_y[idx] = y;
  
       // dbg_printf("Keypoint detected at x = %d, y= %d. idx is %d ," 
         //           "and level is %d, and intensity is %f\n", x, y, idx, i, dk[center]);

        break;
    }

}


void DoG_detector_cuda(Mat img, int **k_x, int **k_y, int *n, float th_c, float th_r, 
                  int levels, float sigma)
{
    
    double start, end;

    /* Device image */
    float *gpu_img;
    float *img_ptr = (float*) img.ptr<float>();

    /* Get width and height */
    int w = img.cols;
    int h = img.rows;
   
    /* BLock width */
    int block_width = BSZ;
 
    /* Calculate image size in bytes */
    size_t img_sz = w * h * sizeof(float);
 
    /* Generate DoG Levels */
    float sigma_l[10];

    for (int i = -1; i < levels - 1; i ++) 
        sigma_l[i+1] = (float) i;

    /* Create device and host filter banks */
    filter_bank fb, fbd;
    create_DoG_bank (&fb, levels, sqrt(2), sigma, sigma_l);

    /* Copy device filter bank to host */
    copy_DoG_bank_device(&fb, &fbd);

    /* Allocate image memory in device */
    hipMalloc(&gpu_img, img_sz);
        
    /* Copy image from host to device */
    hipMemcpy(gpu_img, img_ptr, img_sz, hipMemcpyHostToDevice);

    /* Calculate Grid Size */   
    const dim3 block(block_width, block_width);
    const dim3 grid( (w + block.x - 1) / block.x, (h + block.y - 1) / block.y);

    start = CycleTimer::currentSeconds();

    /* Launch Kernel */
    //DoG_Kernel<<<grid,block>>>(gpu_img, w, h, fbd);

    //DoG_Pyramid<<<grid,block>>>(gpu_img, w, h, fbd);
    vertical_blur<<<grid,block>>>(gpu_img, w, h, fbd);
    hipDeviceSynchronize();
    end = CycleTimer::currentSeconds();
    // cout<<"Filters took "<<end-start<<" seconds"<<endl;
    horizontal_blur<<<grid,block>>>(gpu_img, w, h, fbd);
    hipDeviceSynchronize();
    //end = CycleTimer::currentSeconds();
    //cout<<"Filters took "<<end-start<<" seconds"<<endl;
    start = CycleTimer::currentSeconds();
    DoG_Kernel<<<grid,block>>>(gpu_img, w, h, fbd);
    hipDeviceSynchronize();
    end = CycleTimer::currentSeconds();
    //cout<<"CUDA MINMAX kernel took "<<end-start<<" seconds"<<endl;

    //cout<<"CUDA KERNEL took "<<end-start<<" seconds"<<endl;

    dbg_printf("Finished calling kernel\n");

    /* Free device memory */
    hipFree(gpu_img);

    /* Copy results from device to host */

    hipMemcpyFromSymbol(n, HIP_SYMBOL(kp), sizeof(int));
    dbg_printf("Detected %d keypoints \n",*n);

    *k_y = (int *) malloc(*n * sizeof(int));
    *k_x = (int *) malloc(*n * sizeof(int));
    
    hipMemcpyFromSymbol(*k_x, HIP_SYMBOL(kp_x), sizeof(int)* (*n));
    hipMemcpyFromSymbol(*k_y, HIP_SYMBOL(kp_y), sizeof(int)* (*n));

    /* Clear kp */
    int zero = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(kp), &zero, sizeof(int));
}


